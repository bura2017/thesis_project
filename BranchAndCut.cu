/**
 * Copyright (c) 2016 ISP RAS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "BranchAndCut.h"
#include "TransitionMatrix.h"
#include <fstream>

int num_of_probs;

static void memFree(Matrix &matrix, d_matrix dev_trans) {
  std::cout << num_of_probs << " probs "<< std::endl;
  matrix.freeHost();
  hipFree(dev_trans.e);
}

bool branchAndCut (Matrix &input) {
  num_of_probs = 0;

  static d_matrix dev_trans;
  dev_trans_init(dev_trans, input.cols);
  if (gpuDualSimplexAsync (input, dev_trans) == 0) {
    std::cout << num_of_probs << std::endl;
    return false;
  }

  taskTree *root = new taskTree;
  orderList *start_order = new orderList (root);

  if (root->branchPoint(input)) {
    std::cout << num_of_probs << std::endl;
    return true;
  }

  Matrix matrix (input, input.cols);

  while (1) {
    num_of_probs++;
    if (num_of_probs % 10000 == 0) {
      std::cout << "bz" << std::endl;
    }

    start_order->task->branchTask();

    for (int l = 0; l < NUM_OF_DAUGHT; l++) {
      initMatrix (matrix, input, start_order->task->next[l], dev_trans);
      if (gpuDualSimplexAsync (matrix) == 0) {
        delete start_order->task->next[l];
      } else {
        if (start_order->task->next[l]->branchPoint (matrix)) {
          memFree(matrix, dev_trans);
          return true;
        }
        if (start_order->next == NULL) {
          start_order->next = new orderList (start_order->task->next[l]);
        } else {
          start_order->next = start_order->next->pasteTask(start_order->task->next[l]);
        }
      }
    }
    orderList *temp = start_order;
    start_order = start_order->next;
    delete temp;
    if (start_order == NULL) {
      memFree(matrix, dev_trans);
      return false;
    }

  }
}
