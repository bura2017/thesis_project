/**
 * Copyright (c) 2016 ISP RAS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "MatrixMultip.h"
#include "BranchAndCut.h"
#include "TransitionMatrix.h"
#include <fstream>

int num_of_probs;

static void memFree(Matrix &matrix, d_matrix dev_trans) {
  std::cout << num_of_probs << " probs "<< std::endl;
  matrix.freeHost();
  hipFree(dev_trans.e);
}

void initMatrix(Matrix &matrix, const Matrix &input, taskTree * &task, d_matrix &dev_trans) {
  //std::cout << "Init new matrix..." << std::endl;
  CHECK_NULL(task);
  matrix = input;

  int cut_rows = 0;
  for (taskTree *branch = task; branch->prev != NULL; branch = branch->prev) {
    cut_rows++;
  }
  if (!cut_rows) {
    CHECK_NULL(NULL);
  }
  Matrix cuts (cut_rows, input.cols);
  cut_rows = 0;
  for (taskTree *branch = task; branch->prev != NULL; branch = branch->prev) {
    int point = branch->prev->point;
    double value = branch->prev->value;
    if (branch->num == 0) {
      cuts.e[cut_rows + point * cuts.m] = 1.0;
      cuts.e[cut_rows + 0 * cuts.m] = value;
    } else {
      cuts.e[cut_rows + point * cuts.m] = -1.0;
      cuts.e[cut_rows + 0 * cuts.m] = -(value + 1.0);
    }
    cut_rows++;
  }
  Matrix temp_matrix(cuts.rows, cuts.cols);
  //Matrix check_matrix(cuts.rows, cuts.cols);
  //MatMul(cuts, dev_trans, temp_matrix);                   //35356.4
  cublas_multip(cuts, dev_trans, temp_matrix); //36686.2
  //std::cout << "Check ... " << cuts.rows << std::endl;
  //check_matrix.print("Check.txt");
  /*for (int i = 0; i < temp_matrix.rows * temp_matrix.cols; i++) {
    if (cmp(temp_matrix.e[i], check_matrix.e[i]) != 0) {
      std::cout << i << ' ' << temp_matrix.e[i] << ' ' << check_matrix.e[i] << std::endl;
    }
  }/**/
  matrix.add_cuts(temp_matrix);
}

bool branchAndCut (Matrix &input) {
  num_of_probs = 0;

  static d_matrix dev_trans;
  dev_trans_init(dev_trans, input.cols);
  if (gpuDualSimplexAsync (input, dev_trans) == 0) {
    std::cout << num_of_probs << std::endl;
    return false;
  }

  taskTree *root = new taskTree;
  orderList *start_order = new orderList (root);

  if (root->branchPoint(input)) {
    std::cout << num_of_probs << std::endl;
    return true;
  }

  Matrix matrix (input, input.cols);

  while (1) {
    num_of_probs++;
    if (num_of_probs % 10000 == 0) {
      std::cout << "bz" << std::endl;
    }

    start_order->task->branchTask();

    for (int l = 0; l < NUM_OF_DAUGHT; l++) {
      initMatrix (matrix, input, start_order->task->next[l], dev_trans);
      if (gpuDualSimplexAsync (matrix) == 0) {
        delete start_order->task->next[l];
      } else {
        if (start_order->task->next[l]->branchPoint (matrix)) {
          memFree(matrix, dev_trans);
          return true;
        }
        if (start_order->next == NULL) {
          start_order->next = new orderList (start_order->task->next[l]);
        } else {
          start_order->next = start_order->next->pasteTask(start_order->task->next[l]);
        }
      }
    }
    orderList *temp = start_order;
    start_order = start_order->next;
    delete temp;
    if (start_order == NULL) {
      memFree(matrix, dev_trans);
      return false;
    }

  }
}
