/**
 * Copyright (c) 2016 ISP RAS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "TransitionMatrix.h"
#include "../Epsilon.h"

int dev_trans_init(d_matrix &dev_trans, Matrix &input) {
  const int side = input.cols;
  dev_trans.rows = side;
  dev_trans.cols = side;
  dev_trans.m = side + BLOCK_SIZE - 1;
  CHECK_CUDA(hipMalloc(&dev_trans.e, sizeof(double) * dev_trans.m * dev_trans.m));

  Matrix trans(dev_trans.m, dev_trans.m);
  for (int i = 0; i < dev_trans.m; i++) {
    trans.e[i + i * trans.m] = 1.0;
  }//identity mantrix
  for (int j = 0; j < side; j++) {
    for (int i = 1; i < side; i++) {
      trans.e[i + j * trans.m] = - input.e[i + j * input.m];
      if (cmp(input.e[i + j * input.m], 0.0) == 0) {
        trans.e[i + j * trans.m] = 0.0;
      }
    }
  }
  CHECK_CUDA(hipMemcpy (dev_trans.e, trans.e, sizeof(double) * dev_trans.m * dev_trans.m, hipMemcpyHostToDevice));

  return 0;
}
int dev_trans_free(d_matrix &dev_trans) {
  hipFree(dev_trans.e);
  return 0;
}
