/**
 * Copyright (c) 2016 ISP RAS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DualSimplex.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

static int flag;
static data_async data0, data1;

static void memInit(const int rows, const int cols, int m) {
  flag = 0;
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDeviceProperties (&prop, 0));

  int size = MAX_BLOCKS * m;

  data0.dev_matrix.rows = rows;
  data0.dev_matrix.cols = MAX_BLOCKS;
  data0.dev_matrix.m = m;
  CHECK_CUDA (hipMalloc ((void**)&data0.dev_matrix.e, sizeof(double) * size));

  data1.dev_matrix.rows = rows;
  data1.dev_matrix.cols = MAX_BLOCKS;
  data1.dev_matrix.m = m;
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_matrix.e, sizeof(double) * size));

  CHECK_CUDA (hipMalloc ((void**)&data0.dev_col, sizeof(double) * m));
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_col, sizeof(double) * m));

  CHECK_CUDA(hipStreamCreate(&data0.stream));
  CHECK_CUDA(hipStreamCreate(&data1.stream));

  data0.pin_matrix = new Matrix(rows, MAX_BLOCKS, hipHostMallocDefault, m - rows);
  data1.pin_matrix = new Matrix(rows, MAX_BLOCKS, hipHostMallocDefault, m - rows);
}
static void memFree () {
  hipFree(data0.dev_matrix.e);
  hipFree(data1.dev_matrix.e);

  hipFree(data0.dev_col);
  hipFree(data1.dev_col);

  CHECK_CUDA(hipStreamDestroy(data0.stream));
  CHECK_CUDA(hipStreamDestroy(data1.stream));

  data0.pin_matrix->freeHost();
  data1.pin_matrix->freeHost();
  delete data0.pin_matrix;
  delete data1.pin_matrix;
  //std::cout << flag << std::endl;
}

int gpuDualSimplexAsync (Matrix &matrix) {
  CHECK_NULL(matrix.e);

  memInit(matrix.rows, matrix.cols, matrix.m);

  while (1) {
    flag ++;

    int pivot_row = pivotRow (matrix);
    if (!pivot_row) {
      memFree ();
      return flag;
    }

    int pivot_col = pivotColumn (matrix, pivot_row);
    if (!pivot_col) {
      memFree ();
      return -flag;
    }

    matrixTransformAsync (matrix, pivot_row, pivot_col, data0, data1);
  }
}
