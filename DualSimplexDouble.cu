#include "DualSimplex.h"
#include <iostream>

static int flag;
static data_full_task data0, data1;

static void memInit(Matrix &matrix0, Matrix &matrix1) {
  flag = 0;
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDeviceProperties (&prop, 0));

  int size = MAX_BLOCKS * matrix0.m;

  data0.matrix = &matrix0;
  data0.dev_matrix.rows = matrix0.rows;
  data0.dev_matrix.cols = MAX_BLOCKS;
  data0.dev_matrix.m = matrix0.m;
  CHECK_CUDA (hipMalloc ((void**)&data0.dev_matrix.e, sizeof(double) * size));

  data1.matrix = &matrix1;
  data1.dev_matrix.rows = matrix1.rows;
  data1.dev_matrix.cols = MAX_BLOCKS;
  data1.dev_matrix.m = matrix1.m;
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_matrix.e, sizeof(double) * size));

  CHECK_CUDA (hipMalloc ((void**)&data0.dev_col, sizeof(double) * matrix0.m));
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_col, sizeof(double) * matrix1.m));

  CHECK_CUDA(hipStreamCreate(&data0.stream));
  CHECK_CUDA(hipStreamCreate(&data1.stream));

  data0.pin_matrix = new Matrix(matrix0.rows, MAX_BLOCKS, hipHostMallocDefault, matrix0.m - matrix0.rows);
  data1.pin_matrix = new Matrix(matrix1.rows, MAX_BLOCKS, hipHostMallocDefault, matrix1.m - matrix1.rows);
}
static void memFree () {
  hipFree(data0.dev_matrix.e);
  hipFree(data1.dev_matrix.e);

  hipFree(data0.dev_col);
  hipFree(data1.dev_col);

  CHECK_CUDA(hipStreamDestroy(data0.stream));
  CHECK_CUDA(hipStreamDestroy(data1.stream));

  data0.pin_matrix->freeHost();
  data1.pin_matrix->freeHost();
  delete data0.pin_matrix;
  delete data1.pin_matrix;
  //std::cout << flag << std::endl;
}

int *gpuDualSimplexDouble (Matrix &matrix0, Matrix &matrix1) {
  CHECK_NULL(matrix0.e);
  CHECK_NULL(matrix1.e);

  //std::cout << "//=======================" << std::endl;
  memInit(matrix0, matrix1);
  int *check = new int[2];
  check[0] = 0;
  check[1] = 0;

  while (1) {
    flag ++;
    //std::cout << flag << std::endl;
    if (flag % 1000000000 == 0) {
      std::cout << "ups" << std::endl;
      memFree ();
      return 0;
    }

    if (check[0] == 0) {
      data0.piv_row = pivotRow (matrix0);
      if (data0.piv_row == 0) {
        check[0] = flag;
      }
    }

    if (check[1] == 0) {
      data1.piv_row = pivotRow (matrix1);
      if (data1.piv_row == 0) {
        check[1] = flag;
      }
    }

    if (check[0] == 0) {
      data0.piv_col = pivotColumn (matrix0, data0.piv_row);
      if (data0.piv_col == 0) {
        check[0] = -flag;
      }
    }

    if (check[1] == 0) {
      data1.piv_col = pivotColumn (matrix1, data1.piv_row);
      if (data1.piv_col == 0) {
        check[1] = -flag;
      }
    }

    //std::cout << flag << ' ' << data1.piv_row << ' ' << data1.piv_col << std::endl;

    if (check[0] == 0 && check[1] == 0) {
      matrixTransformDouble (data0, data1);
    }
    if (check[0] == 0 && check[1] != 0) {
      data_async data_0, data_1;
      data_0 = data0;
      data_1 = data1;
      matrixTransformAsync (*data0.matrix, data0.piv_row, data0.piv_col, data_0, data_1);
    }
    if (check[0] != 0 && check[1] == 0) {
      data_async data_0, data_1;
      data_0 = data0;
      data_1 = data1;
      matrixTransformAsync (*data1.matrix, data1.piv_row, data1.piv_col, data_0, data_1);
    }
    if (check[0] != 0 && check[1] != 0) {
      memFree();
      return check;
    }
  }
}
