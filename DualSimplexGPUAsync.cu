#include "DualSimplex.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

static int flag;
static data_async data0, data1;

static void memInit(const int rows, const int cols, int m) {
  flag = 0;
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDeviceProperties (&prop, 0));

  int size = MAX_BLOCKS * m;

  data0.dev_matrix.rows = rows;
  data0.dev_matrix.cols = MAX_BLOCKS;
  data0.dev_matrix.m = m;
  CHECK_CUDA (hipMalloc ((void**)&data0.dev_matrix.e, sizeof(double) * size));

  data1.dev_matrix.rows = rows;
  data1.dev_matrix.cols = MAX_BLOCKS;
  data1.dev_matrix.m = m;
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_matrix.e, sizeof(double) * size));

  CHECK_CUDA (hipMalloc ((void**)&data0.dev_col, sizeof(double) * m));
  CHECK_CUDA (hipMalloc ((void**)&data1.dev_col, sizeof(double) * m));

  CHECK_CUDA(hipStreamCreate(&data0.stream));
  CHECK_CUDA(hipStreamCreate(&data1.stream));

  data0.pin_matrix = new Matrix(rows, MAX_BLOCKS, hipHostMallocDefault, m - rows);
  data1.pin_matrix = new Matrix(rows, MAX_BLOCKS, hipHostMallocDefault, m - rows);
}
static void memFree () {
  hipFree(data0.dev_matrix.e);
  hipFree(data1.dev_matrix.e);

  hipFree(data0.dev_col);
  hipFree(data1.dev_col);

  CHECK_CUDA(hipStreamDestroy(data0.stream));
  CHECK_CUDA(hipStreamDestroy(data1.stream));

  data0.pin_matrix->freeHost();
  data1.pin_matrix->freeHost();
  delete data0.pin_matrix;
  delete data1.pin_matrix;
  //std::cout << flag << std::endl;
}

int gpuDualSimplexAsync (Matrix &matrix) {
  CHECK_NULL(matrix.e);

  //std::cout << "===========simplex============" << std::endl;
  memInit(matrix.rows, matrix.cols, matrix.m);

  while (1) {
    flag ++;
    if (flag % 1000000000000 == 0) {
      std::cout << "ups" << std::endl;
      //memFree ();
      //return 0;
    }

    int pivot_row = pivotRow (matrix);
    if (!pivot_row) {
      memFree ();
      return flag;
    }

    int pivot_col = pivotColumn (matrix, pivot_row);
    if (!pivot_col) {
      memFree ();
      return -flag;
    }
//std::cout << flag << ' ' << pivot_row << ' ' << pivot_col << std::endl;

    matrixTransformAsync (matrix, pivot_row, pivot_col, data0, data1);
  }
}
