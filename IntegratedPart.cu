#include "Matrix.h"
#include "HandleError.h"
#include "BranchAndCut.h"
#include "CudaDeviceProperties.h"
#include "TimeCuda.h"
#include "GenTest.h"
#include "Epsilon.h"

#include <cstdlib>
#include <iostream>
#include <cstring>

int flag;

int main (int argc, char **argv) {
  hipSetDevice(0);
  hipDeviceReset();
  CHECK_CUDA (hipSetDeviceFlags (hipDeviceMapHost));
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDeviceProperties (&prop, 0));

  if (!prop.deviceOverlap) {
    std::cout << "Device will not handle overlaps, so no speed up from streams" << std::endl;
  }
  if (prop.integrated) {
    std::cout << "Integrated device gives speed up from zero-copy memory" << std::endl;
  }

  int test_num = 0;
  flag = time(NULL);
  //std::cout << flag << std::endl;

  while (test_num < 10) {
    const int vars = 20;
    const int ineqs = 30;

    //gen_test(test_num, vars, ineqs, flag);

    char filename[MAX_LENG];
    //strcpy (filename, test_files[test_num]);
    sprintf(filename, "Vars-%d_Ineqs-%d_%d.ilp", vars, ineqs, test_num);
    //std::cout << filename << std::endl;
    //std::cin >> filename;
    if (filename[0] == '0') {
      return 0;
    }

    char fullname[MAX_LENG] = "Input.txt";
    //sprintf(fullname, "/home/valerius/cuda-workspace/Benchmarks_txt/TestGenerator/Type1/%s", filename);
    //sprintf(fullname, "TestGenerator/%s", filename);

    Matrix input(fullname);

    cuda_time time;
    cmp ((double) input.cols, (double) input.rows);

    //Simplex testing
    {
      int iters_man = 22757;
      {
        time.start();
        Matrix matrix(input);
        int iters_cpu = cpuDualSimplex (matrix);
        time.stop();
        iters_man = iters_cpu;
        std::cout << iters_man << std::endl;
        std::cout << time.time() << std::endl;
        if (iters_cpu != iters_man) {
          std::cout << iters_cpu << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer cpu" << std::endl;
        }
        if (flag < 0) {
          std::cout << "ERROR wrong answer" << std::endl;
        }
        std::cout << "check " << checkCorrect(input, matrix) << std::endl;
      }/**/

      /*{
        time.start();
        Matrix matrix(input);
        int iters_async = gpuDualSimplexAsync (matrix);
        time.stop();
        std::cout << time.time() << std::endl;
        //std::cout << " GPU async time " << time.time() << std::endl;
        if (iters_async != iters_man) {
          std::cout << iters_async << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer async" << std::endl;
        }
      }/**/

      /*{
        time.start();
        Matrix matrix0(input);
        Matrix matrix1(input);
        int *iters_double = gpuDualSimplexDouble (matrix0, matrix1);
        time.stop();
        std::cout << time.time() << std::endl;
        std::cout << " GPU double result " << iters_double[0] << ' ' << iters_double[1] << std::endl;
        if (iters_double[0] != iters_man) {
          std::cout << iters_double[0] << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer double" << std::endl;
        }
        if (iters_double[1] != iters_man) {
          std::cout << iters_double[1] << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer double" << std::endl;
        }
        delete [] iters_double;
      }/**/

      /*{
        time.start();
        Matrix matrix(input);
        d_matrix trans_matrix;
        trans_matrix.rows = matrix.cols;
        trans_matrix.cols = matrix.cols;
        trans_matrix.m = matrix.cols;
        CHECK_CUDA(hipMalloc (&trans_matrix.e, sizeof(double) * trans_matrix.m * trans_matrix.cols));
        int iters_async = gpuDualSimplexAsync (matrix, trans_matrix);
        hipFree(trans_matrix.e);
        time.stop();
        std::cout << time.time() << std::endl;
        //std::cout << " GPU async time " << time << std::endl;
        if (iters_async != iters_man) {
          std::cout << iters_async << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer async" << std::endl;
        }
      }/**/

      /*{
        time.start();
        Matrix matrix(input);
        int iters_sync = gpuDualSimplexSync (matrix);
        time.stop();
        std::cout << time.time() << std::endl;
        if (iters_sync != iters_man) {
          std::cout << iters_sync << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer sync" << std::endl;
        }
      }/**/

      /*{
        time.start();
        Matrix matrix(input);
        int iters_sync_dev = gpuDualSimplexSyncDev (matrix);
        time.stop();
        std::cout << time.time() << std::endl;
        if (iters_sync_dev != iters_man) {
          std::cout << iters_sync_dev << " != " << iters_man << std::endl;
          std::cout << "ERROR wrong answer sync dev" << std::endl;
        }
      }/**/

    }

    //BranchAndCut testing

    /*{
      Matrix matrix(input);
      time.start();
      std::cout << (branchAndBound(matrix) ? "sat" : "unsat");
      time.stop();
      std::cout << " time " << time.time() << std::endl << std::endl;
    }/**/
    //test_num++;
    return 0;
  }

}
