#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016 ISP RAS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "MatrixMultip.h"

__global__
void multip(d_matrix left, d_matrix right, d_matrix answ) {
  int row = threadIdx.x;
  int col = threadIdx.y;

  int block_rows = blockDim.x;
  int block_cols = blockDim.y;

  int box = row + col * block_rows;

  __shared__ double left_box[BLOCK_SIZE * BLOCK_SIZE + 1];
  __shared__ double right_box[BLOCK_SIZE * BLOCK_SIZE + 1];

  double val = 0.0;

  int bound = gridDim.y;
  int a_col = col, b_row = row;
  int a_row = row + blockIdx.x * block_rows, b_col = col + blockIdx.y * block_cols;
  for (int m = 0; m < bound; m++) {
    left_box[box] = left.e[a_row + a_col * left.m];
    right_box[box] = right.e[b_row + b_col * right.m];

    __syncthreads();

    for (int e = 0; e < block_rows; e++) {
      val += left_box[row + e * block_rows] * right_box[e + col * block_rows];
    }
    a_col += block_cols;
    b_row += block_rows;
    __syncthreads();
  }
  answ.e[a_row + b_col * answ.m] = val;
}

int MatMul(const Matrix &cuts, const d_matrix dev_trans, Matrix &result) {
  int side = (cuts.rows > BLOCK_SIZE ? BLOCK_SIZE : cuts.rows);
  dim3 dimBlock(side, side);

  d_matrix temp;
  temp.rows = cuts.rows;
  temp.cols = cuts.cols;
  temp.m = cuts.rows;
  size_t size = sizeof(double) * cuts.rows * cuts.cols;
  CHECK_CUDA(hipMalloc(&temp.e, size));
  CHECK_CUDA(hipMemcpy(temp.e, cuts.e, size, hipMemcpyHostToDevice));

  d_matrix d_cuts;
  d_cuts.rows = (cuts.rows - 1) / side + 1;
  d_cuts.cols = (cuts.cols - 1) / side + 1;
  dim3 dimGrid(d_cuts.rows, d_cuts.cols);
  d_cuts.cols *= side;
  d_cuts.rows *= side;
  d_cuts.m = cuts.rows;
  size = sizeof(double) * d_cuts.rows * d_cuts.cols;
  hipMalloc(&d_cuts.e, size);
  iden_matr<<<d_cuts.cols,d_cuts.rows>>> (d_cuts);
  copyMatrix<<<temp.cols,temp.rows>>>(d_cuts,temp);
  d_cuts.rows = cuts.rows;
  d_cuts.cols = cuts.cols;

  d_matrix d_res;
  d_res.rows = d_cuts.rows;
  d_res.cols = d_cuts.cols;
  d_res.m = d_cuts.m;
  hipMalloc(&d_res.e, size);

  multip<<<dimGrid, dimBlock>>>(d_cuts, dev_trans, d_res);

  copyMatrix<<<cuts.cols,cuts.rows>>>(temp, d_res);
  size = sizeof(double) * cuts.rows * cuts.cols;
  hipMemcpy(result.e, temp.e, size, hipMemcpyDeviceToHost);

  hipFree(d_res.e);
  hipFree(d_cuts.e);
  hipFree(temp.e);

  return 0;
}
