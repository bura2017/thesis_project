
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hiprand.h>
#include <hipblas.h>
#include "HandleError.h"
#include "MatrixMultip.h"

void cublas_multip (Matrix const &left, Matrix const &right, Matrix &answ) {
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);//default
  hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED);// not allowed id default

  d_matrix dev_left;
  dev_left.rows = left.rows;
  dev_left.cols = left.cols;
  dev_left.m = left.m;
  CHECK_CUDA(hipMalloc(&dev_left.e, sizeof (float) * dev_left.m * dev_left.cols));
  CHECK_CUBLAS(hipblasSetMatrix(left.rows, left.cols, sizeof(float), left.e, left.m,
      dev_left.e, dev_left.m));

  d_matrix dev_right;
  dev_right.rows = right.rows;
  dev_right.cols = right.cols;
  CHECK_CUDA(hipMalloc(&dev_right.e, sizeof (float) * dev_right.rows * dev_right.cols));
  CHECK_CUBLAS(hipblasSetMatrix(right.rows, right.cols, sizeof(float), right.e, right.rows,
      dev_right.e, dev_right.rows));

  d_matrix dev_answ;
  dev_answ.rows = answ.rows;
  dev_answ.cols = answ.cols;
  dev_answ.m = answ.m;
  CHECK_CUDA(hipMalloc(&dev_answ.e, sizeof (float) * dev_answ.m * dev_answ.cols));
  float a = 1.0, b = 0.0;

  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dev_left.rows, dev_right.cols, dev_left.cols,
      &a, dev_left.e, dev_left.m, dev_right.e, dev_right.m, &b, dev_answ.e, dev_answ.m));

  CHECK_CUBLAS(hipblasGetMatrix(answ.rows, answ.cols, sizeof(float), dev_answ.e, dev_answ.rows, answ.e, answ.rows));

  hipblasDestroy(handle);
  hipFree(dev_left.e);
  hipFree(dev_right.e);
  hipFree(dev_answ.e);

}
void cublas_multip (Matrix const &left, d_matrix const &dev_right, Matrix &answ) {
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);//default
  hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED);// not allowed id default

  d_matrix dev_left;
  dev_left.rows = left.rows;
  dev_left.cols = left.cols;
  dev_left.m = left.m;
  CHECK_CUDA(hipMalloc(&dev_left.e, sizeof (float) * dev_left.m * dev_left.cols));
  CHECK_CUBLAS(hipblasSetMatrix(left.rows, left.cols, sizeof(float), left.e, left.m,
      dev_left.e, dev_left.m));

  d_matrix dev_answ;
  dev_answ.rows = answ.rows;
  dev_answ.cols = answ.cols;
  dev_answ.m = answ.m;
  CHECK_CUDA(hipMalloc(&dev_answ.e, sizeof (float) * dev_answ.m * dev_answ.cols));
  float a = 1.0, b = 0.0;

  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dev_left.rows, dev_right.cols, dev_left.cols,
      &a, dev_left.e, dev_left.m, dev_right.e, dev_right.m, &b, dev_answ.e, dev_answ.m));

  CHECK_CUBLAS(hipblasGetMatrix(answ.rows, answ.cols, sizeof(float), dev_answ.e, dev_answ.rows, answ.e, answ.rows));

  hipblasDestroy(handle);

  hipFree(dev_left.e);
  hipFree(dev_answ.e);
}
void cublas_multip (d_matrix &dev_left, d_matrix &dev_right, d_matrix &dev_answ, hipStream_t stream) {
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);//host is default
  hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED);// not allowed is default

  float a = 1.0, b = 0.0;

  hipblasSetStream(handle, stream);
  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dev_left.rows, dev_right.cols, dev_left.cols,
      &a, dev_left.e, dev_left.m, dev_right.e, dev_right.m, &b, dev_answ.e, dev_answ.m));

  hipblasDestroy(handle);

}
void cublas_multip (d_matrix &dev_left, d_matrix &dev_right, d_matrix &dev_answ) {
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);//host is default
  hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED);// not allowed is default

  float a = 1.0, b = 0.0;

  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dev_left.rows, dev_right.cols, dev_left.cols,
      &a, dev_left.e, dev_left.m, dev_right.e, dev_right.m, &b, dev_answ.e, dev_answ.m));

  hipblasDestroy(handle);

}
