#include "MatrixTransformation.h"


int matrixTransformSync(Matrix &matrix, const int row, const int col, d_matrix &dev_matrix, float *dev_col) {
  float div = - matrix.e[row + col * matrix.m];
  for (int i = 0; i < matrix.rows; i++) {
    matrix.e[i + col * matrix.m] /= div;
  }
  CHECK_CUDA (hipMemcpy (dev_matrix.e, matrix.e, sizeof (float) * matrix.m * matrix.cols,
      hipMemcpyHostToDevice));
  CHECK_CUDA (hipMemcpy (dev_col, &matrix.e[0 + col * matrix.m], sizeof (float) * matrix.rows,
      hipMemcpyHostToDevice));

  matrixTransform<<<matrix.cols, matrix.rows>>>(dev_matrix, row, dev_col);

  CHECK_CUDA (hipMemcpy (matrix.e, dev_matrix.e, sizeof (float) * matrix.m * matrix.cols,
      hipMemcpyDeviceToHost));

  for (int j = 0; j < matrix.cols; j++) {
    matrix.e[row + j * matrix.m] = (j == col ? -1.0 : 0.0);
  }
  return 0;
}
